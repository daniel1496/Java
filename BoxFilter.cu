
#include "hip/hip_runtime.h"

#include <vector>
#include <stdio.h>
//#include "lodepng.h"
#include <png.h>

#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdarg.h>

#define PNG_DEBUG 3
#include <png.h>

/**
 * Because program need to read and write png file, so need to use libpng.
 * Below code of read/write png file is referenced from link http://zarb.org/~gc/html/libpng.html
 * I modify it a little bit to suitable for using CUDA
 */

png_byte color_type;
png_byte bit_depth;

png_structp png_ptr;
png_infop info_ptr;
int number_of_passes;
png_bytep * row_pointers;


void read_png_file(const char* filename, unsigned char * &pImage, unsigned &width, unsigned & height)
{
	char header[8];    // 8 is the maximum size that can be checked
	int y;

	/* open file and test for it being a png */
	FILE *fp = fopen(filename, "rb");

	fread(header, 1, 8, fp);


	/* initialize stuff */
	png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

	info_ptr = png_create_info_struct(png_ptr);


	setjmp(png_jmpbuf(png_ptr));


	png_init_io(png_ptr, fp);
	png_set_sig_bytes(png_ptr, 8);

	png_read_info(png_ptr, info_ptr);

	width = png_get_image_width(png_ptr, info_ptr);
	height = png_get_image_height(png_ptr, info_ptr);
	color_type = png_get_color_type(png_ptr, info_ptr);
	bit_depth = png_get_bit_depth(png_ptr, info_ptr);

	number_of_passes = png_set_interlace_handling(png_ptr);
	png_read_update_info(png_ptr, info_ptr);

	setjmp(png_jmpbuf(png_ptr));

	row_pointers = (png_bytep*) malloc(sizeof(png_bytep) * height);
	for (y=0; y<height; y++)
			row_pointers[y] = (png_byte*) malloc(png_get_rowbytes(png_ptr,info_ptr));

	png_read_image(png_ptr, row_pointers);

	fclose(fp);

	// setup continuous 1D array
	pImage = (unsigned char *) malloc(width * height * 4);
	for (y = 0; y < height; y++)
		memcpy(pImage + y * (width *4), row_pointers[y], width *4);

}


void write_png_file(const char* filename, unsigned char * pImage, unsigned width, unsigned height) {

	int y;
	FILE *fp = fopen(filename, "wb");

	/* initialize stuff */
	png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

	info_ptr = png_create_info_struct(png_ptr);

	setjmp(png_jmpbuf(png_ptr));

	png_init_io(png_ptr, fp);


	/* write header */
	setjmp(png_jmpbuf(png_ptr));

	png_set_IHDR(png_ptr, info_ptr, width, height,
				 bit_depth, color_type, PNG_INTERLACE_NONE,
				 PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

	png_write_info(png_ptr, info_ptr);


	/* write bytes */
	setjmp(png_jmpbuf(png_ptr));


	for (y = 0; y < height; y++)
		memcpy(row_pointers[y], pImage + y * (width *4), width *4);

	png_write_image(png_ptr, row_pointers);


	/* end write */
	setjmp(png_jmpbuf(png_ptr));

	png_write_end(png_ptr, NULL);

	/* cleanup heap allocation */
	for (y=0; y<height; y++)
			free(row_pointers[y]);
	free(row_pointers);

	fclose(fp);
}


/**
 * Box filter function use CPU only
 */
void boxFilterCPU(unsigned char * pImageIn, unsigned char * pImageOut, int width, int height, int kernelSize)
{
	int kernelRadius = kernelSize / 2;
	for (int y = 0; y < height; y++)
		for (int x = 0; x < width; x++)
		{
			unsigned sumR = 0;
			unsigned sumG = 0;
			unsigned sumB = 0;
			unsigned count = 0;

			for (int k_y = -kernelRadius; k_y <= kernelRadius; k_y++)
			{
				for (int k_x = -kernelRadius; k_x <= kernelRadius; k_x++)
				{

					if ((y + k_y >= 0 && y + k_y < height) && (x + k_x >= 0 && x + k_x < width))
					{
						count++;
						sumR += pImageIn[4 * width * (y + k_y) + 4 * (x + k_x) + 0];
						sumG += pImageIn[4 * width * (y + k_y) + 4 * (x + k_x) + 1];
						sumB += pImageIn[4 * width * (y + k_y) + 4 * (x + k_x) + 2];
					}
				}
			}

			pImageOut[4 * width * y + 4 * x + 0] = sumR / count;
			pImageOut[4 * width * y + 4 * x + 1] = sumG / count;
			pImageOut[4 * width * y + 4 * x + 2] = sumB / count;
			pImageOut[4 * width * y + 4 * x + 3] = pImageIn[4 * width * y + 4 * x + 3];
		}
}


/**
 * Box filter kernel function
 */
__global__ void boxFilterKernel(unsigned char * pImageIn, unsigned char * pImageOut, unsigned width, unsigned height, int kernelRadius)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;    // compute index x,y of pixel processed by current thread
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if ( x < width && y < height)			// make sure x,y are not out of image range, because index of x,y can be larger than image range
	{
		unsigned int sumR = 0;		    // sum of RED value of neightbourhood of pixel at x,y, incldue pixel x,y
		unsigned int sumG = 0;			// sum of GREEN value of neightbourhood of pixel at x,y, incldue pixel x,y
		unsigned int sumB = 0;			// sum of BLUE value of neightbourhood of pixel at x,y, incldue pixel x,y
		unsigned int count = 0;        // number of neighbourhood of pixel at x,y, include pixel x,y

		for (int k_y = -kernelRadius; k_y <= kernelRadius; k_y++)		// loop to all neighbourhood pixel
		{
			for (int k_x = -kernelRadius; k_x <= kernelRadius; k_x++)
			{

				if ((y + k_y >= 0 && y + k_y < height) && (x + k_x >= 0 && x + k_x < width))		// make sure neightbourhood pixel still in image's range
				{
					count++;
					sumR += pImageIn[4 * width * (y + k_y) + 4 * (x + k_x) + 0];		// sum up all neighbourhood value
					sumG += pImageIn[4 * width * (y + k_y) + 4 * (x + k_x) + 1];
					sumB += pImageIn[4 * width * (y + k_y) + 4 * (x + k_x) + 2];
				}
			}
		}

		// Update output value for all channels at pixel x,y
		pImageOut[4 * width * y + 4 * x + 0] = (unsigned char) (sumR / count);
		pImageOut[4 * width * y + 4 * x + 1] = (unsigned char) (sumG / count);
		pImageOut[4 * width * y + 4 * x + 2] = (unsigned char) (sumB / count);
		pImageOut[4 * width * y + 4 * x + 3] = pImageIn[4 * width * y + 4 * x + 3];   // Update Alpha value from input image.
	}
}

/**
 * Box filter function use GPU
 */
void boxFilterGPU(unsigned char * pImageIn, unsigned char * pImageOut, unsigned width, unsigned height, unsigned kernelSize)
{
	unsigned char * dev_pImageOut;        // Declare buffer of input image on device
	unsigned char * dev_pImageIn;		  // Declare buffer of output image on device

	int numElement = width * height * 4;  // compute size of image in order to allocate memory on device

	// Allocate device memory on GPU
	hipMalloc((void**)& dev_pImageIn, numElement);			// Allocate memory on device for input image
	hipMalloc((void**)& dev_pImageOut, numElement);        // Allocate memory on device for output image

	// Transfer data from CPU to GPU
	hipMemcpy(dev_pImageIn, pImageIn, numElement, hipMemcpyHostToDevice);    // Copy input image from CPU to GPU memory.

	int kernelRadius = kernelSize / 2;
	// Setup grid, block size to launch kernel
	const int blockSizeX = 32;
	const int blockSizeY = 32;
	dim3 blocks(blockSizeX, blockSizeY);   // Image width and height will divide into blocks of 32x32 pixels


	dim3 grid(width / blockSizeX + 1, height / blockSizeY + 1);     // Compute number of blocks

	boxFilterKernel<<<grid, blocks>>>(dev_pImageIn, dev_pImageOut, width, height, kernelRadius);  // Launch kernel for processing

	// Transfer result data from device to host
	hipMemcpy(pImageOut, dev_pImageOut, numElement, hipMemcpyDeviceToHost);

	// Free cuda device memory
	hipFree(dev_pImageIn);
	hipFree(dev_pImageOut);
}


int main(int argc, char *argv[])
{
	unsigned char * pImage;			// buffer to store input image
	unsigned char * pImageOut;		// buffer to store output image
	unsigned width;					// image width
	unsigned height;				// image height

	read_png_file(argv[1], pImage, width, height);   		// read png file into input buffer


	pImageOut = new unsigned char[width * height * 4];		// Allocate buffer of output image

	unsigned kernelSize = 5;								// kernel size, can change this to any positive odd number

	boxFilterGPU(pImage, pImageOut, width, height, kernelSize);		//call box filtering function, can use CPU or GPU function to compare speed.

	write_png_file("Output.png", pImageOut, width, height);	// write png file of output buffer


	delete pImage;											// Cleanup memory after finnishing
	delete pImageOut;

    return 0;
}

