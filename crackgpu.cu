
#include "hip/hip_runtime.h"

#include <vector>
#include <stdio.h>

#define ENCRYPTED_TEXT_LEN 10
#define PLAIN_TEXT_LEN 4
#define ALPHABET_NUM 26

__device__ void CudaCrypt(char* rawPassword, char * newPassword) {

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;

	for (int i = 0; i < 10; i++) {
		if (i >= 0 && i < 6) { //checking all lower case letter limits
			if (newPassword[i] > 122) {
				newPassword[i] = (newPassword[i] - 122) + 97;
			} else if (newPassword[i] < 97) {
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		} else { //checking number section
			if (newPassword[i] > 57) {
				newPassword[i] = (newPassword[i] - 57) + 48;
			} else if (newPassword[i] < 48) {
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
}


/**
 *  Kernel function for cracking app
 */
__global__ void crack_kernel(char *encrypted_text, char* plain_text)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	char candidate_text[4];  		// candidate plain text
	char new_encrypted[10];			// new encrypted code from candidate_text

	if (x < 26 && y < 26 && z < 100)		// each threads check 1 combination, make sure combination is valid
	{
			// Convert x,y,z value to candidate text
			candidate_text[0] = 97 + x;				// 'a' has ascii value is 97, x from 0-> 25, so 97 + x can be from 'a' --> 'z'
			candidate_text[1] = 97 + y;
			candidate_text[2] = 48 + (z / 10);      // '0' has ascii value is 48, so this hold first number from '0' --> '9'
			candidate_text[3] = 48 + (z % 10);		// compute second number

			CudaCrypt(candidate_text, new_encrypted);		// encrypt candidate text to new_encrypted

			bool match = true;
			for (int i = 0; i < 10; i++)		// Check if new_encrypted is match with encrypted_text
			{
				if (encrypted_text[i] != new_encrypted[i])  // if any character is different, will not check any more
				{
					match = false;
					break;
				}
			}

			if (match)			// if a solution found, copy solution to result buffer
			{
				plain_text[0] = candidate_text[0];
				plain_text[1] = candidate_text[1];
				plain_text[2] = candidate_text[2];
				plain_text[3] = candidate_text[3];
			}
	}
}

void crack_gpu(char *encrypted_text){
	printf("Cracking encrypted code %s\n", encrypted_text);

	char * dev_encrypted_text;			// declare memory device to store encrypted text
	char * dev_plain_text;				// declare memory device to store result plain text

	char result_text[5];

	// Allocate device memory
	hipMalloc((void**)&dev_encrypted_text, ENCRYPTED_TEXT_LEN);
	hipMalloc((void**)&dev_plain_text, PLAIN_TEXT_LEN);

	// Transfer encrypted text to device to cracking
	hipMemcpy(dev_encrypted_text, encrypted_text, ENCRYPTED_TEXT_LEN, hipMemcpyHostToDevice);

	dim3 block(8, 8, 16);												// There are 26 x 26 x 100 combinations need to check
																		// These are divide into smaller block of 8x8x16
	dim3 grid(ALPHABET_NUM / 8 + 1, ALPHABET_NUM / 8 + 1, 100 / 16 + 1 );	// Declare number of blocks need to cover all combination

	crack_kernel<<<grid,block>>>(dev_encrypted_text, dev_plain_text); // Launch cracking kernel

	// Transfer crack result from device to host
	hipMemcpy(result_text, dev_plain_text, PLAIN_TEXT_LEN, hipMemcpyDeviceToHost);
	result_text[4] = '\0';

	printf("Cracking result string %s\n", result_text);  // Display cracked result

	// Free device memory
	hipFree(dev_encrypted_text);
	hipFree(dev_plain_text);

}

#define FILE_NAME "encrypted_password.bin"

int main(int argc, char *argv[]){

    char * filename;                // File name contain encrypted text
    char *encryped_text = NULL;     // encrypted text
    int fsize = 0;                  // size of file, it also length of encrypted text
    FILE *fp;                       // pointer to file

    filename = argv[1];
    fp = fopen(filename, "r");      // open file in reading mode
    if(fp) {
        fseek(fp, 0, SEEK_END);     // move file cursor to end of file
        fsize = ftell(fp);          // get size of file
        rewind(fp);                 // set file cursor to beginning of file, ready for reading

        encryped_text = (char*) malloc(sizeof(char) * fsize);        // Allocate buffer to store content of file
        fread(encryped_text, 1, fsize, fp);                          // read content of file to buffer

        fclose(fp);                                                  // Close file
    }
  crack_gpu(encryped_text);

  free(encryped_text);

  return 0;
}


